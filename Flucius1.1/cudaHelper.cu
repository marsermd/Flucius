#include "cudaHelper.h"

int getThreads (int runCount) {
	return runCount > THREADS_CNT ? THREADS_CNT : runCount;
}

dim3 getBlocks (int runCount) {
	runCount = (runCount - 1) / THREADS_CNT + 1;
	int blocksx = 1;
	int blocksy = 1;
	if (runCount > 65535) {
		blocksx = 65535;
		blocksy = (runCount - 1) / 65535 + 1;
	} else {
		blocksx = runCount;
	}
	return dim3(blocksx, blocksy);
}

void checkCudaErrors(hipError_t cudaStatus) {
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda failed %s\n", hipGetErrorString(cudaStatus));
	}
}

void checkCudaErrorsWithLine(char *message) {
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda failed. Message:\n%s\n erroro:\n%s", message, hipGetErrorString(cudaStatus));
	}
}