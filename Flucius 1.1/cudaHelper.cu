#include "cudaHelper.h"

int getThreads(int runCount, int threads_cnt) {
	return runCount > threads_cnt ? threads_cnt : runCount;
}

dim3 getBlocks(int runCount, int threads_cnt) {
	runCount = (runCount - 1) / threads_cnt + 1;
	int blocksx = 1;
	int blocksy = 1;
	if (runCount > 65535) {
		blocksx = 65535;
		blocksy = (runCount - 1) / 65535 + 1;
	} else {
		blocksx = runCount;
	}
	return dim3(blocksx, blocksy);
}

void checkCudaErrors(hipError_t cudaStatus) {
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda failed %s\n", hipGetErrorString(cudaStatus));
	}
}

void checkCudaErrorsWithLine(char *message) {
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda failed. Message:\n%s\n erroro:\n%s", message, hipGetErrorString(cudaStatus));
	}
}